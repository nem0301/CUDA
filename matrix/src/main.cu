#include <stdio.h>
#include <memory.h>

#include <Matrix.cu>


int main()
{
	int size = 1024;
	int size2 = 1024;
	float data[size * size2];

	//int k[] = { 2, 2, 3, 4 };
	for (int i = 0; i < size2; i++)
	{
		for (int j = 0; j < size; j++)
		{
			int x = size * i + j;
			data[x] = i;
		}
	}

	Matrix mat1(data, size, size2);

	//int t[] = { 5, 6, 7, 8 };
	for (int i = 0; i < size2; i++)
	{
		for (int j = 0; j < size; j++)
		{
			int x = size * i + j;
			data[x] = i;
		}
	}

	Matrix mat2(data, size, size2);
	Matrix mat3(size, size2);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	mat1.inverse();

	

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapse;
	hipEventElapsedTime(&elapse, start, stop);
	printf("%.6f ms\n", elapse);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	//mat1.printMatrix();
	//mat2.printMatrix();
	//mat3.printMatrix();
	

	return 0;
}
